#include "hip/hip_runtime.h"
#include <random>
#include <algorithm>
#include "reaction_diffusion.h"
#include "cuda_helpers.h"
#include "color.h"
#include <stdlib.h>

const std::vector<Color> TURBO({
    Color(48, 18, 59),
    Color(62, 55, 144),
    Color(69, 90, 205),
    Color(70, 123, 243),
    Color(62, 155, 254),
    Color(41, 187, 236),
    Color(25, 214, 204),
    Color(33, 234, 172),
    Color(70, 248, 132),
    Color(120, 254, 90),
    Color(163, 253, 60),
    Color(196, 241, 52),
    Color(225, 221, 55),
    Color(246, 195, 58),
    Color(254, 165, 49),
    Color(251, 128, 34),
    Color(240, 91, 18),
    Color(222, 61, 8),
    Color(196, 37, 3),
    Color(163, 18, 1),
    Color(122, 4, 3)
});

ReactionDiffusion::ReactionDiffusion(int _nx, int _ny, float _dt, float _du, float _dv, float _k, float _f) :
    Automaton<float>(_nx, _ny), dt(_dt), du(_du), dv(_dv), f(_f), k(_k) {

    // Allocate CUDA arrays
    CUDA_CHECK( hipMalloc((void**) &d_state_u,     stateSize) );
    CUDA_CHECK( hipMalloc((void**) &d_state_v,     stateSize) );
    CUDA_CHECK( hipMalloc((void**) &d_nextState_u, stateSize) );
    CUDA_CHECK( hipMalloc((void**) &d_nextState_v, stateSize) );
}

Color ReactionDiffusion::getColor(float stateVal) {
    float t = std::clamp((stateVal -0.3f) / 0.7f, 0.0f, 1.0f);
    int ind = static_cast<int>((TURBO.size() - 1) * t);
    return TURBO.at(ind);
}

__global__
void kernel_diffuse(float *nextState, float *state, float D, float dt, int nx, int ny) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float dy = 2.5f / ny;
    float dx = 2.5f / nx;

    float x_scale = 1.0f / (dx * dx);
    float y_scale = 1.0f / (dy * dy);

    if (i < nx && j < ny) {
        int id = wrapIndex2D(i, j, nx, ny);
        int n  = wrapIndex2D(i, j+1, nx, ny);
        int s  = wrapIndex2D(i, j-1, nx, ny);
        int e  = wrapIndex2D(i+1, j, nx, ny);
        int w  = wrapIndex2D(i-1, j, nx, ny);
        nextState[id] = state[id] + dt * D * (
            (state[e] + state[w] - 2 * state[id]) * x_scale +
            (state[n] + state[s] - 2 * state[id]) * y_scale
        );
    }
}

__global__
void kernel_react(float *U, float*V, float k, float f, float dt, int nx, int ny) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < nx && j < ny) {
        int id = wrapIndex2D(i, j, nx, ny);
        float u = U[id];
        float v = V[id];
        U[id] += dt * (-u*v*v + f * (1 - u));
        V[id] += dt * (u*v*v - (f + k) * v);
    }
}

void ReactionDiffusion::update() {

    if (tick == 0) {
        // Create initial condition
        std::default_random_engine generator;
        std::uniform_real_distribution<float> distribution(-0.1, 0.1);
        std::vector<float> conc_v(nx * ny, 0);

        for (int j = 0; j < ny; j++) {
            for (int i = 0; i < nx; i++) {
                int id = i + j * nx;
                if (float(i) / nx > 0.45 && float(i) / nx < 0.55 && float(j) / ny > 0.45 && float(j) / ny < 0.55) {
                    state[id] = 0.5;
                    conc_v[id] = 0.25;
                } else {
                    state[id] = 1.0;
                    conc_v[id] = 0.0;
                }

                float r1 = (float)rand()/(float)(RAND_MAX);
                float r2 = (float)rand()/(float)(RAND_MAX);

                state[id]  *= (1 + 1.0 * (r1-0.5));
                conc_v[id] *= (1 + 1.0 * (r2-0.5));
            }
        }

        std::cout << "System initialized." << std::endl;

        // Copy initial condition to GPU
        CUDA_CHECK( hipMemcpy(d_state_u, state.data(),  stateSize, hipMemcpyHostToDevice) );
        CUDA_CHECK( hipMemcpy(d_state_v, conc_v.data(), stateSize, hipMemcpyHostToDevice) );
    }

    for (int i = 0; i < 10; i++) {
        const int BLOCK_SIZE_X = 32;
        const int BLOCK_SIZE_Y = 32;

        // Perform diffusion of both components
        dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
        dim3 gridSize(nx / BLOCK_SIZE_X, ny / BLOCK_SIZE_Y, 1);
        kernel_diffuse<<<gridSize, blockSize>>>(d_nextState_u, d_state_u, du, dt, nx, ny);
        kernel_diffuse<<<gridSize, blockSize>>>(d_nextState_v, d_state_v, dv, dt, nx, ny);

        // swap previous and new states
        std::swap(d_state_u, d_nextState_u);
        std::swap(d_state_v, d_nextState_v);

        // Reactions
        kernel_react<<<gridSize, blockSize>>>(d_state_u, d_state_v, k, f, dt, nx, ny);
    }

    // Copy GPU memory to CPU
    CUDA_CHECK( hipMemcpy(state.data(), d_state_u, stateSize, hipMemcpyDeviceToHost) );


    tick += 5;
}

ReactionDiffusion::~ReactionDiffusion() {
    CUDA_CHECK( hipFree(d_state_u) );
    CUDA_CHECK( hipFree(d_state_v) );
    CUDA_CHECK( hipFree(d_nextState_u) );
    CUDA_CHECK( hipFree(d_nextState_v) );
}

